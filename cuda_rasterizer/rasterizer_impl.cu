#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
// TODO: 进一步继续理解这部分代码
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
	const float* orig_points,
	const float* viewmatrix,
	const float* projmatrix,
	bool* present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps. 
// Run once per Gaussian (1:N mapping).
// 计算2d高斯椭圆中心点points_xy在2d像素平面上占据的tile的tileID，
// 并将tileID|depth组合成64位的key值，value值为高斯球的编号
__global__ void duplicateWithKeys(
	int P,									// 高斯球数量
	const float2* points_xy,
	const float* depths,
	const uint32_t* offsets,				// 累计的tiles数量的数组
	uint64_t* gaussian_keys_unsorted,		// 未排序的key
	uint32_t* gaussian_values_unsorted,		// 未排序的value
	int* radii,
	dim3 grid)								// block编号的上限
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		//第idx个高斯球前面已经占据的tiles总数
		// 前序和计算的结果
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		// 计算像素点points_xy[idx]在半径为radii[idx]的圆所占据的网格编号的最小值和最大值
		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a 
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values 
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth. 
		// 通过这样的编码，可以让所有高斯球先通过tile进行sort，然后通过depth进行sort

		// 遍历该高斯球涉及的所有tiles，并存到gaussian_keys_unsorted和
		// gaussian_values_unsorted两个数组
		// 这两个数组之前已经使用每一个高斯占用的网格数量进行初始化内存分配过，
		// 所以每一个高斯球只要offset计算对了这边的填入也就会是对的
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				// 计算当前block所在的tile ID
				uint64_t key = y * grid.x + x;
				// 左移32位，空出后面32位
				key <<= 32;
				// 后面32位存放该高斯的深度值
				key |= *((uint32_t*)&depths[idx]);
				// 记录高斯key，用于后面的排序
				gaussian_keys_unsorted[off] = key;
				// 记录高斯的id信息
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

// Check keys to see if it is at the start/end of one tile's range in 
// the full sorted list. If yes, write start/end of this tile. 
// Run once per instanced (duplicated) Gaussian ID.
// 看函数名字以及注释，大概就是去从所有的list里面去找每一个tile对应的value的信息
// 并行的是num_rendered，也就是所有的tile数量
// 会遍历所有的tile。uint2就相当于一个pair
// ranges：每一项存储对应tile的的id范围[0，L-1]，
// 这个id表示的是在point_list_keys中的索引，通过binningState.point_list找到对应高斯球编号

// 每个线程都会执行这段逻辑，但是对应不同的 idx 值，所以每个线程负责处理不同的元素。
// CUDA会自动地并行执行这些线程，每个线程独立地更新 ranges 数组的不同位置，
// 因此不需要显式地对 idx 进行递增操作。

// 函数最后的目的：ranges中的key对应每一个tile的ID，然后value是这个tile在point list中的范围，来检索到高斯球的情况
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, uint2* ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	// 本质上idx就像是for循环中的i，这边就是在检索。因为每一个idx都会对应一个key，所以可以并行去做
	uint64_t key = point_list_keys[idx];
	// 读当前tile 的id
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		// 如果发现不等了，就说明进入到新的区域了
		// 这个时候就记录一下这个上一个tile（也就是之前）的end的id
		// 以及下一个新的tile id开始的位置
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
		// 如果相等就无事发生，
	}
	// 到最后了，就记录最后的 结束的位置就是L。不过为啥不是L-1
	if (idx == L - 1)
		ranges[currtile].y = L;
}

// Mark Gaussians as visible/invisible, based on view frustum testing
void CudaRasterizer::Rasterizer::markVisible(
	int P,
	float* means3D,
	float* viewmatrix,
	float* projmatrix,
	bool* present)
{
	checkFrustum << <(P + 255) / 256, 256 >> > (
		P,
		means3D,
		viewmatrix, projmatrix,
		present);
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char*& chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	//参数：第三个in，第四个out，最后一个num。
	// 当第一个参数为NULL时, 所需的分配大小被写入第二个参数，并且不执行任何工作
    // https://github.com/dmlc/cub/blob/master/cub/device/device_scan.cuh
	// 源码中写的很清楚，先determine，后分配，这边就写到scan size中区了
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char*& chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N, 128);
	obtain(chunk, img.n_contrib, N, 128);
	obtain(chunk, img.ranges, N, 128);
	return img;
}

// memory allocation
CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char*& chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char* (size_t)> geometryBuffer,
	std::function<char* (size_t)> binningBuffer,
	std::function<char* (size_t)> imageBuffer,
	const int P, int D, int M,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* opacities,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* cam_pos,
	const float tan_fovx, float tan_fovy,
	const bool prefiltered,
	float* out_color,
	int* radii,
	bool debug)
{
	// 从fov解算 fx fy
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	// P就是三维高斯点的数量
	// 根据 GeometryState 的大小和点的数量 P 来计算所需的内存块大小
	size_t chunk_size = required<GeometryState>(P);
	// 调用rasterize_points.cu文件中的resizeFunctional函数里面
	// 嵌套的匿名函数lambda来调整显存块大小，并返回首地址
	char *chunkptr = geometryBuffer(chunk_size);
	// 分配各个内存位置,申请显存
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	// 计算当前图像的大小涵盖的tile的范围
	// 图像块数量
	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	// 块中线程数量 16*16
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	// 模版函数required调用fromChunk函数来获取内存，返回结束地址，也即所需存储大小
	size_t img_chunk_size = required<ImageState>(width * height);
	char* img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
		P, D, M,
		means3D,
		(glm::vec3*)scales,
		scale_modifier,
		(glm::vec4*)rotations,
		opacities,
		shs,
		geomState.clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, projmatrix,
		(glm::vec3*)cam_pos,
		width, height,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		radii,
		geomState.means2D,
		geomState.depths,
		geomState.cov3D,
		geomState.rgb,
		geomState.conic_opacity,
		tile_grid,
		geomState.tiles_touched,
		prefiltered
	), debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	// 把指针（point_offsets + P - 1），也就是point_offsets数组的最后一个元素的值，
	// 赋给num_rendered，也就是总共覆盖的tiles数量
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	// 计算所需的BinningState的数量，即每个高斯球覆盖的tile都有对应的装箱状态BinningState数据
	size_t binning_chunk_size = required<BinningState>(num_rendered);
	// 调整显存块大小，返回首地址
	char* binning_chunkptr = binningBuffer(binning_chunk_size);
	// 利用显存块首地址作为参数，调用fromchunk申请显存
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key 
	// and corresponding dublicated Gaussian indices to be sorted
	// 16*16 = 256,每一个块一共有256个线程
	duplicateWithKeys << <(P + 255) / 256, 256 >> > (
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,					// 累计高斯球touch的tile数目
		binningState.point_list_keys_unsorted,		// 存储key [tile|depth]
		binningState.point_list_unsorted,			// 存储对应的高斯球id
		radii,
		tile_grid)									// 全图块数量
	CHECK_CUDA(, debug)

	// 找到最高位
	int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	// 32+bit表示了用于排序的位数
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
		binningState.list_sorting_space,	// 辅助空间
		binningState.sorting_size,			// 辅助空间的大小
		binningState.point_list_keys_unsorted, binningState.point_list_keys,
		binningState.point_list_unsorted, binningState.point_list,
		num_rendered, 0, 32 + bit), debug) // 排序操作涉及的tile数量（所有tile总数），开始bit位，结束bit位

	// 将内存区域imgState.ranges数据设置为0
	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges << <(num_rendered + 255) / 256, 256 >> > (
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// Let each tile blend its range of Gaussians independently in parallel
	// 每一个tile自己render自己的
	const float* feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	CHECK_CUDA(FORWARD::render(
		tile_grid, block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		geomState.means2D,
		feature_ptr,
		geomState.conic_opacity,
		imgState.accum_alpha,
		imgState.n_contrib,
		background,
		out_color), debug)

	return num_rendered;
}

// Produce necessary gradients for optimization, corresponding
// to forward render pass

// backward的入口函数，主流程
// incoming gradient --- dL_dpixels
void CudaRasterizer::Rasterizer::backward(
	const int P, int D, int M, int R,
	const float* background,
	const int width, int height,
	const float* means3D,
	const float* shs,
	const float* colors_precomp,
	const float* scales,
	const float scale_modifier,
	const float* rotations,
	const float* cov3D_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const float* campos,
	const float tan_fovx, float tan_fovy,
	const int* radii,
	char* geom_buffer,
	char* binning_buffer,
	char* img_buffer,
	const float* dL_dpix,
	float* dL_dmean2D,
	float* dL_dconic,
	float* dL_dopacity,
	float* dL_dcolor,
	float* dL_dmean3D,
	float* dL_dcov3D,
	float* dL_dsh,
	float* dL_dscale,
	float* dL_drot,
	bool debug)
{
	GeometryState geomState = GeometryState::fromChunk(geom_buffer, P);
	BinningState binningState = BinningState::fromChunk(binning_buffer, R);
	ImageState imgState = ImageState::fromChunk(img_buffer, width * height);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	const dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	const dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Compute loss gradients w.r.t. 2D mean position, conic matrix,
	// opacity and RGB of Gaussians from per-pixel loss gradients.
	// If we were given precomputed colors and not SHs, use them.
	const float* color_ptr = (colors_precomp != nullptr) ? colors_precomp : geomState.rgb;
	CHECK_CUDA(BACKWARD::render(
		tile_grid,
		block,
		imgState.ranges,
		binningState.point_list,
		width, height,
		background,
		geomState.means2D,
		geomState.conic_opacity,
		color_ptr,
		imgState.accum_alpha,
		imgState.n_contrib,
		dL_dpix,
		(float3*)dL_dmean2D,
		(float4*)dL_dconic,
		dL_dopacity,
		dL_dcolor), debug)

	// Take care of the rest of preprocessing. Was the precomputed covariance
	// given to us or a scales/rot pair? If precomputed, pass that. If not,
	// use the one we computed ourselves.
	const float* cov3D_ptr = (cov3D_precomp != nullptr) ? cov3D_precomp : geomState.cov3D;
	CHECK_CUDA(BACKWARD::preprocess(P, D, M,
		(float3*)means3D,
		radii,
		shs,
		geomState.clamped,
		(glm::vec3*)scales,
		(glm::vec4*)rotations,
		scale_modifier,
		cov3D_ptr,
		viewmatrix,
		projmatrix,
		focal_x, focal_y,
		tan_fovx, tan_fovy,
		(glm::vec3*)campos,
		(float3*)dL_dmean2D,
		dL_dconic,
		(glm::vec3*)dL_dmean3D,
		dL_dcolor,
		dL_dcov3D,
		dL_dsh,
		(glm::vec3*)dL_dscale,
		(glm::vec4*)dL_drot), debug)
}