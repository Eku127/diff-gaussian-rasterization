#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
// 计算投影的cov，公式5
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	// 这边会使用transpose来变更不同的存储方式存储下来的矩阵
	
	// 1. 计算相机系下的中心点
	// 从世界坐标系变换到相机坐标系
	// 计算t的目的是为了计算Jacobian
	// 
	//	viewmatrix = | Rcw tcw |
	//				 | 0   1   |
	// u(3x1) --> t(3x1)
	float3 t = transformPoint4x3(mean, viewmatrix);

	// 2. 计算这个点下展开的Jacobian J
	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	// Equation (3) in "Math Gsplat"
	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	// 3. 计算Rcw，world to camera矩阵
	// 本质上是取出旋转
	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	// U = J * W
	glm::mat3 T = W * J;

	// 4. 计算sigma'
	// sigma'(cov) = U * sigma * U^T
	// 写作sigma，实际是sigma^T
	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	// 相当于上面的公式左右同时转置
	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
// reparametrization
// 此处计算3D空间上的cov，通过s和r
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	// TODO: Rotation的存储方式？xyzw还是wxyz，按照代码来看是后者
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	// 没毛病，数学上相当于存了个R^T
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	// row oriented
	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	// 计算这个点在不在clip space里面
	// 按照计算的逻辑基本上是来者不拒
	// 是通过camera space下的z来计算的
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	// projmatrix是直接可以将世界系点变换到NDC系下的，包含了View的变换
	// gaussian_renderer/__init__.py 中对projmatrix赋值
	// scene/cameras.py 中对Viewmat和projmat进行计算
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	// 通过S和R计算cov3D, 因为是对称矩阵，所以记录六个数即可
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		// Get the cov3D indicator
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	// 计算2D的投影
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	// 计算cov_inv, 也就是这边写的conic
	// 
	// conic = 1 / det * | c   -b |
	// 					 | -b  a  |
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// Compute extent in screen space (by finding eigenvalues of
	// 2D covariance matrix). Use extent to compute a bounding rectangle
	// of screen-space tiles that this Gaussian overlaps with. Quit if
	// rectangle covers 0 tiles. 
	// 求解特征值
	// det(cov - lambda * I) = 0
	// (c - lambda) * (a - lambda) - b^2 = 0
	// 求解得到俩lambda
	float mid = 0.5f * (cov.x + cov.z);
	float lambda1 = mid + sqrt(max(0.1f, mid * mid - det));
	float lambda2 = mid - sqrt(max(0.1f, mid * mid - det));
	// 按照文章所述，用3*lambda作为判定区域
	float my_radius = ceil(3.f * sqrt(max(lambda1, lambda2)));
	// 求解uv坐标
	// TODO: p_proj应该是相机坐标系下的点，这边不应该需要将点变换一次到ND系下去？
	// p_view才是相机坐标系下的点，p_proj是NDC坐标系下的点
	// projmatrix直接从世界系变换NDC系
	// 之前就已经除以tw了
	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };

	// 通过uv坐标求解2D高斯椭圆覆盖的tile id
	uint2 rect_min, rect_max;
	getRect(point_image, my_radius, rect_min, rect_max, grid);
	// 
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	// 存放的内容，output的内容
	// 点的相机系下的z
	depths[idx] = p_view.z;
	// 投影点的半径大小
	radii[idx] = my_radius;
	// 投影点的uv
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = { conic.x, conic.y, conic.z, opacities[idx] };
	// touch了多少
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
// 限制每个线程块的最大线程数量为 BLOCK_X * BLOCK_Y
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color)
{
	// Identify current tile and associated min/max pixel range.
	// 创建线程块
	auto block = cg::this_thread_block();
	// 水平方向的线程块数目
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	// 最大最小像素坐标
	// block.group_index() ---- 当前block在grid中的索引，blockIdx
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	// 当前thread对应的像素点在图象上坐标
	// block.thread_index() ---- 代表当前thread在block中的索引，threadIdx
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	// 二维坐标变成一维坐标后像素的id
	uint32_t pix_id = W * pix.y + pix.x;
	// as float
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	// 找到当前tile对应的高斯信息，基于之前的计算结果
	// block.group_index() 当前tile的id
	// 当前tile对应的高斯球id list
	// number of Gaussians for this tile.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	// 每一个线程需要计算的大概高斯球数量
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	// 当前tile对应的高斯球数量
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	// 同一个block(tile)中共享的内容
	// 各个线程处理的高斯球编号id
	__shared__ int collected_id[BLOCK_SIZE];
	// ~ 2d平面投影坐标
	__shared__ float2 collected_xy[BLOCK_SIZE];
	// ~ 2d协方差的逆以及不透明度
	__shared__ float4 collected_conic_opacity[BLOCK_SIZE];

	// Initialize helper variables
	// 透射率
	float T = 1.0f;
	// 计算经过了多少高斯
	uint32_t contributor = 0;
	// 最终经过的高斯球数量
	uint32_t last_contributor = 0;
	// 最后渲染的颜色
	float C[CHANNELS] = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		// 如果一个block里面全部thread完成，则退出循环
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		// 对于同一个block，把所有的高斯球全部存到shared内存中，这样方便后续的使用
		// 因此这个block中的每一个线程都会直接从shared mem中获取得到这个block对应的gaussian的信息
		// thread_rank：当前线程在组内的标号，区间为[0, num_threads)
		int progress = i * BLOCK_SIZE + block.thread_rank();
		// 当前线程有效，高斯球不越界
		if (range.x + progress < range.y)
		{
			// 当前线程处理的高斯球编号
			int coll_id = point_list[range.x + progress];
			// 当前线程高斯球处理的id
			collected_id[block.thread_rank()] = coll_id;
			// 当前线程高斯球2d中心坐标
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			// 协方差+不透明度
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		// 这是一个 CUDA 中的内置函数，用于在当前线程块内进行同步操作。
		// 它会阻塞调用该函数的所有线程，直到该线程块内的所有线程都执行到了这个同步点为止
		block.sync();

		// Iterate over current batch
		// 每一个线程遍历高斯球
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			// 高斯球计数
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			// 通过当前pixf点到这个高斯点的距离解算概率密度函数
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			// 如果alpha太小就不要
			if (alpha < 1.0f / 255.0f)
				continue;
			// 通过alpha计算Transmittance
			float test_T = T * (1 - alpha);
			// T太小就跳过，并且认为就是结束了
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			// 解算color
			// Every Gaussian gets added onto every pixel (color += color * alpha * T) 
			// until a pixel gets fully rendered, i.e., T is close to 0:
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			// 记录
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	// (P + 255) / 256 calculates the number of blocks
	// needed based on the total number of elements P and 
	// the desired number of threads per block
	// 256, specifies the number of threads per block.
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}